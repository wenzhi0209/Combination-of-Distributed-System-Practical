#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>

int main2(void)
{
    //@@ generate random data serially
    thrust::host_vector<int> h_vec(200);
    std::generate(h_vec.begin(), h_vec.end(), rand);

    //@@ transfer to device and compute sum
    thrust::device_vector<int> d_vec = h_vec;

    //Parallel Vector Addition
    int x = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::plus<int>());

    //@@ Display the sum
    std::cout << x << std::endl;

    return 0;
}