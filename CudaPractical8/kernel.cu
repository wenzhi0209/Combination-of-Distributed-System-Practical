﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>

int main1()
{
    // generate 100 random numbers serially
    thrust::host_vector<int> h_vec(100);
    std::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;

    // sort data on the device 
    thrust::sort(d_vec.begin(), d_vec.end());

    //@@ transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(),h_vec.begin());

    // print h_vec
    for (int i = 0; i < h_vec.size(); i++)
        std::cout << "_vec[" << i << "] = " << h_vec[i] << std::endl;

    return 0;
}

