#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#define THREADS_PER_BLOCK 128

void matrixMultiplyCPU(float *a, float *b, float *c, int width) {
     float result;

     for (int row = 0; row < width; row++) {
          for (int col = 0; col < width; col++) {
               result = 0;
               for (int k = 0; k < width; k++) {
                    result += a[row * width + k] * b[k * width + col];
               }
               c[row * width + col] = result;
          }
     }
}

__global__ void matrixMultiplySimple(float *a, float *b, float *c, int width) {
     
    int col = threadIdx.x + blockIdx.x + blockDim.x;
    int row = threadIdx.y + blockIdx.y + blockDim.y;

     float result = 0;

     if (col < width && row < width) {
          for (int k = 0; k < width; k++) {
               result += a[row * width + k] * b[k * width + col];
          }
          c[row * width + col] = result;
     }
}

int main() {
     int width = 200; // Define width of square matrix
                      // Initialise grid and block variables
     int sqrtThreads = sqrt(THREADS_PER_BLOCK);
     int nBlocks = width / sqrtThreads;
     if (width % sqrtThreads != 0) { // Add an extra block if necessary
          nBlocks++;
     }
     dim3 grid(nBlocks, nBlocks, 1);
     dim3 block(sqrtThreads, sqrtThreads, 1); // Max number of threads per block

                                              // Initialise host pointers (dynamically allocated memory) and device pointers
     float *a_h;
     float *b_h;
     float *c_h; // GPU results
     float *d_h; // CPU results
     float *a_d;
     float *b_d;
     float *c_d;

     int size; // Number of bytes required by arrays

               // Create timer
     hipEvent_t start;
     hipEvent_t stop;
     float elapsed1, elapsed2, elapsed3;

     // Print out information about blocks and threads
     printf("Number of threads: %i (%ix%i)\n", block.x*block.y, block.x, block.y);
     printf("Number of blocks: %i (%ix%i)\n", grid.x*grid.y, grid.x, grid.y);

     // Dynamically allocate host memory
     size = width * width * sizeof(float);

     a_h = (float*)malloc(size);
     b_h = (float*)malloc(size);
     c_h = (float*)malloc(size);
     d_h = (float*)malloc(size);

     // Load host arrays with data
     for (int i = 0; i < width; i++) {
          for (int j = 0; j < width; j++) {
               a_h[i * width + j] = i;
               b_h[i * width + j] = i;
          }
     }

     //@@ Allocate device memory
     

     //@@ Copy host memory to device memory
     

     // Start timer for GPU
     hipEventCreate(&start);
     hipEventCreate(&stop);
     hipEventRecord(start, 0);

     //@@ Launch kernel
     

     // Stop timer
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&elapsed1, start, stop);

     // Print execution time
     printf("Time to calculate results on GPU: %f ms\n", elapsed1);

     // Copy results to host
    

     // Start timer for CPU
     hipEventRecord(start, 0);

     // Launch CPU code
     matrixMultiplyCPU(a_h, b_h, d_h, width);

     // Stop timer
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&elapsed2, start, stop);

     // Print execution time
     printf("Time to calculate results on CPU: %f ms\n", elapsed2);

     // Compare results
     for (int i = 0; i < width*width; i++) {
          if (c_h[i] != d_h[i]) {
               printf("Error: CPU and GPU results do not match\n");
               break;
          }
     }


     //@@ Free memory
    

     hipEventDestroy(start);
     hipEventDestroy(stop);

     return 0;
}
